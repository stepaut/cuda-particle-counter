#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <stdio.h>
#include <iostream>


// Count of points in 2D area
const unsigned COUNT = 1000;
// Begin of axises
const int MIN = 0;
// End of axises
const int MAX = 1000;
// Size of checking block
const unsigned SIZE = 100;
// Bins per axis
const int BINS = (MAX - MIN) / SIZE;


void generateArray(unsigned seed, int* a) {
	for (int i = 0; i < COUNT; i++) {
		srand(seed * (i+1) + i * i);
		a[i] = rand() % (MAX - MIN) + MIN;
	}
}

__global__ void wherePoint(int* x, int* y, unsigned *res) {
	// --- The number of threads does not cover all the data size
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	while (i < COUNT) {
		int binX = -1;
		int binY = -1;

		char c = 0;

		for (int k = 0; k < BINS; k++) {
			if (MIN + k * SIZE <= x[i] && x[i] <= MIN + (k + 1) * SIZE) {
				binX = k;
				c++;
			}

			if (MIN + k * SIZE <= y[i] && y[i] <= MIN + (k + 1) * SIZE) {
				binY = k;
				c++;
			}

			if (c == 2) {
				break;
			}
		}

		atomicAdd(&res[binX * BINS + binY], 1);
		i += offset;
	}
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t runCuda(int* x, int* y, unsigned* res)
{
	int* dev_x = 0;
	int* dev_y = 0;
	unsigned* dev_res = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&dev_res, BINS * BINS * sizeof(unsigned));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_x, COUNT * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_y, COUNT * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_x, x, COUNT * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_y, y, COUNT * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	wherePoint << <1, COUNT >> > (dev_x, dev_y, dev_res);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(res, dev_res, BINS * BINS * sizeof(unsigned), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_res);
	hipFree(dev_x);
	hipFree(dev_y);

	return cudaStatus;
}

int main() {
	int x[COUNT];
	generateArray(4213, x);
	int y[COUNT];
	generateArray(7028, y);

	int dx[COUNT];
	generateArray(9038, dx);
	int dy[COUNT];
	generateArray(1001, dy);

	unsigned res[BINS * BINS] = {};

	clock_t start, stop;

	start = clock();
	hipError_t cudaStatus = runCuda(x, y, res);
	stop = clock();
	float elapsedTime = (float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f;
	printf("Time to generate (CPU): %3.1f ms\n", elapsedTime);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "run failed!");
		return 1;
	}

	for (int i = 0; i < BINS; i++) {
		for (int j = 0; j < BINS; j++) {
			std::cout << res[i * BINS + j];
			std::cout << "\t";
		}
		std::cout << "\n";
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}